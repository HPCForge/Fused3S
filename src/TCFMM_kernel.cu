#include "hip/hip_runtime.h"
#include "config.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <mma.h>
#include <sstream>
#include <stdio.h>
// #include <thrust/device_vector.h>
// #include <thrust/host_vector.h>
// #include <thrust/sort.h>
// #include <thrust/unique.h>
#include <torch/extension.h>
#include <vector>
#include "ptx.h"

using namespace nvcuda;

union Half2Uint32 {
    half2 h2;
    uint32_t u32;
};

struct Scheduler {
  int ind = blockIdx.x;
  int targetRw;

  __device__ bool 
  next_iter(const int* sortedRowWindows, int nRw){
    if(ind < nRw){
      targetRw = sortedRowWindows[ind];
      ind += gridDim.x;
      return true;
    }
    else{
      return false;
    }
  }
};

// Assume each warp has a 8x8 fp16 matrix in row-major order distributed among threads
// Each thread starts with 2 consecutive fp16 values as a half2 (val)
// This function redistributes elements among threads 
// so that val becomes 2 consecutive fp16 values in column-major order
__device__ void shflTransposeWarp(half2 &val, int laneId){
  int col = laneId/4;
  int row = (laneId%4)*2;
  half2 temp[2];
  temp[0] = __shfl_sync(0xffffffff, val, row*4 + col/2);
  temp[1] = __shfl_sync(0xffffffff, val, (row+1)*4 + col/2);
  if((laneId/4)%2 == 0){
    val.x = temp[0].x;
    val.y = temp[1].x;
  }
  else{
    val.x = temp[0].y;
    val.y = temp[1].y;
  }
}

__device__ void saveSddmmResult(float* sum, float* sddmm_result, int tcbId, int last_block){
  int warpId = threadIdx.y;    // warp_index handling multi-dimension > 16.
  int laneId = threadIdx.x; // lanid of each warp.
  // have warp 0 load sum into sddmm_result, which is block row major
  if(warpId == 0){
    int offset = tcbId * BLK_M * BLK_N;
    for(int i = 0; i < 2; i++){ // 2 16x8 blocks
      if(!last_block || i == 0){
        for(int j = 0; j < 2; j++){ // 2 8x8 blocks in each 16x8 block
          int sum_offset = i*BLK_M*BLK_N + j*BLK_N*BLK_N + laneId*2;
          sddmm_result[offset + sum_offset] = sum[sum_offset];
          sddmm_result[offset + sum_offset + 1] = sum[sum_offset + 1];
        }
      }
    }
  }
}

// produce the max value for every subwarp of 4 threads
__device__ void reduceMax(float& max, volatile int laneId){
  // Round 1: Compare with neighbor at offset 1
  max = fmaxf(max, __shfl_xor_sync(0xF << (laneId/4)*4, max, 1, 4));
  // Round 2: Compare with neighbor at offset 2
  max = fmaxf(max, __shfl_xor_sync(0xF << (laneId/4)*4, max, 2, 4));
}

// sum up "sum" for every 4 consecutive threads in a warp
// results is only valid for the first thread in the warp
__device__ void reduceSum(float& sum){
  // offset = 1
  sum += __shfl_down_sync(0xFFFFFFFF, sum, 1, 4);
  // offset = 2
  sum += __shfl_down_sync(0xFFFFFFFF, sum, 2, 4);
}

// deprecated, replaced by storePartialSumShm and addPartialSums
// use atomicAdd to sum up the result of SDDMM.
__device__ void sumWarpI(int i, const uint64_t* tcbBitMap, float* sum, float* D_frag, int tcbId, bool lastBlock, int laneId){
  uint64_t bitMask = 1ULL << (63 - laneId*2);
  uint64_t bitMaskNext = 1ULL << (63 - laneId*2-1);
  if(!lastBlock || i == 0){
    int sumOffset = i*BLK_M*BLK_N;
    for(int j=0; j< 2; j++){// 2 8x8 blocks in each 16x8 block
      if((tcbBitMap[(tcbId+i)*2+j] & bitMask) != 0){
        atomicAdd(&sum[sumOffset + j*BLK_N*BLK_N + laneId*2], D_frag[j*2]);
      }
      if((tcbBitMap[(tcbId+i)*2+j] & bitMaskNext) != 0){
        atomicAdd(&sum[sumOffset + j*BLK_N*BLK_N + laneId*2 + 1], D_frag[j*2 + 1]);
      }
    }
  }
}

//sum should be BLK_M*BLK_M*number of warps.
__device__ void storePartialSumShm(int i, float* sum, float* D_frag, int warpId, int laneId){
  int sumOffset = warpId*BLK_M*BLK_M + i*BLK_M*BLK_N + laneId*2;
  sum[sumOffset] = D_frag[0];
  sum[sumOffset + 1] = D_frag[1];
  sum[sumOffset + BLK_N*BLK_N] = D_frag[2];
  sum[sumOffset + BLK_N*BLK_N + 1] = D_frag[3];
}

__device__ void addPartialSums(float* sum, int tcbId, int tid, int n_warps, const uint64_t* TCblock_bit_map, bool last_block){
  for(int ind = tid; ind < BLK_M*BLK_M; ind += blockDim.x*blockDim.y){
    int block_id = ind / 64; // which 8x8 block ind belongs to
    if(!last_block || block_id < 2){
      int block_offset = ind % 64; // which element in the 8x8 block ind belongs to
      uint64_t bit_mask = 1ULL << (63 - block_offset);
      if((TCblock_bit_map[tcbId*2+block_id] & bit_mask) != 0){
        //skip first warp because it's already loaded into sum
        for(int i = 1; i < n_warps; i++){
          sum[ind] += sum[i*BLK_M*BLK_M + ind];
        }
      }
      else{
        sum[ind] = 0.0f;
      }
    }
    else{
      sum[ind] = 0.0f;
    }
  }
}

__device__ void setQFrag(volatile uint32_t* Q_frag, uint64_t* Q, int bid, int warpId, int laneId, int numNodes, int embeddingDim){
   // Threads of a warp for fetching a 16X16 block of Q.
  // DOC: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html?highlight=wmma#matrix-fragments-for-mma-m16n8k16-with-floating-point-type
  // Here I'm swapping columns of Q to make the memory access more coalesced. 
  // So when loading K, we have to swap the rows accordingly in order to get the same result.
  int rowIdx = bid * BLK_M + laneId/4;
  // /4 because half4.
  int colIdx = warpId * BLK_K/4 + (laneId%4);
  if(rowIdx < numNodes){
    uint64_t val = Q[rowIdx * embeddingDim/4 + colIdx];
    Q_frag[0] = static_cast<uint32_t>(val & 0xFFFFFFFFull);
    Q_frag[2] = static_cast<uint32_t>(val >> 32);
  }
  if(rowIdx + 8 < numNodes){
    uint64_t val = Q[(rowIdx+8) * embeddingDim/4 + colIdx];
    Q_frag[1] = static_cast<uint32_t>(val & 0xFFFFFFFFull);
    Q_frag[3] = static_cast<uint32_t>(val >> 32);
  }
}

__device__ void setQFragShm(float* dynShm, uint64_t* Q, int bid, int warpId, int laneId, int numNodes, int embeddingDim){
  uint32_t* Q_frag = reinterpret_cast<uint32_t*>(dynShm + BLK_M*2 + warpId*blockDim.x*4);
  int rowIdx = bid * BLK_M + laneId/4;
  // /4 because half4.
  int colIdx = warpId * BLK_K/4 + (laneId%4);
  if(rowIdx < numNodes){
    uint64_t val = Q[rowIdx * embeddingDim/4 + colIdx];
    Q_frag[laneId] = static_cast<uint32_t>(val & 0xFFFFFFFFull);
    Q_frag[blockDim.x*2 + laneId] = static_cast<uint32_t>(val >> 32);
  }
  if(rowIdx + 8 < numNodes){
    uint64_t val = Q[(rowIdx+8) * embeddingDim/4 + colIdx];
    Q_frag[blockDim.x + laneId] = static_cast<uint32_t>(val & 0xFFFFFFFFull);
    Q_frag[blockDim.x*3 + laneId] = static_cast<uint32_t>(val >> 32);
  }
}

// __global__ void TC_fusedMM_cuda_kernel(
// 	const int *__restrict__ TCblock_rowid, 		 // offset of each row window.
// 	const uint8_t *__restrict__ TCblocktile_id,  // id of each TC block nonzero element.
// 	const int *__restrict__ TCblock_offset,      // colid of each TC block nonzero element.
// 	const int *__restrict__ sparse_AToX_idx,     // colid of each TC block nonzero element.
// 	const int numNodes, const int numEdges,
// 	const int embeddingDim,    // embedding dimension.
// 	float *__restrict__ in_mat, // input feature matrix.
// 	float *output,              // aggreAGNNed output feature matrix.
// 	torch::Half *edgeAttention, // result of SDDMM.
// 	bool save_edge_attention
// );
// __global__ void TC_fusedMM_fp32_inter_cuda_kernel(
// 	const int *__restrict__ TCblock_rowid, 		 // offset of each row window.
// 	const uint8_t *__restrict__ TCblocktile_id,  // id of each TC block nonzero element.
// 	const int *__restrict__ TCblock_offset,      // colid of each TC block nonzero element.
// 	const int *__restrict__ sparse_AToX_idx,     // colid of each TC block nonzero element.
// 	const int numNodes, const int numEdges,
// 	const int embeddingDim,    // embedding dimension.
// 	torch::Half *__restrict__ in_mat, // input feature matrix.
// 	float *output,              // aggreAGNNed output feature matrix.
// 	float *edgeAttention, // result of SDDMM.
// 	bool save_edge_attention
// );
#if BLK_M == 8 && BLK_N == 32 && BLK_K == 16
__global__ void TC_fusedMM_fp32_inter_m8n32k16_cuda_kernel(
		const int *__restrict__ TCblock_rowid, 		 // offset of each row window.
		const uint8_t *__restrict__ TCblocktile_id,  // id of each TC block nonzero element.
		const int *__restrict__ TCblock_offset,      // colid of each TC block nonzero element.
		const int *__restrict__ sparse_AToX_idx,     // colid of each TC block nonzero element.
		const int numNodes, const int numEdges,
		const int embeddingDim,    // embedding dimension.
		torch::Half *__restrict__ in_mat, // input feature matrix.
		float *output,              // aggreAGNNed output feature matrix.
		float *edgeAttention, // result of SDDMM.
		bool save_edge_attention);
#endif

__global__ void f3sKernel1tb1tcb(
		const int *__restrict__ rowWindowOffset, 		 // offset of each row window.
		const int *__restrict__ sparseAToXidx,     // colid of each TC block nonzero element.
    const uint64_t *__restrict__ tcbBitMap,
    const int numNodes,
    const int embeddingDim,
		torch::Half *__restrict__ Q, 
    torch::Half *__restrict__ K, 
    torch::Half *__restrict__ V,
		float *output,              // output feature matrix.
		float *sddmmResult, // result of SDDMM
    bool applySoftmax
);

__global__ void f3sKernel1tb1rw(
    const int *__restrict__ rowWindowOffset,
    const int *__restrict__ sparseAToXidx, 
    const uint64_t *__restrict__ tcbBitMap,
    int embeddingDim,
    ulonglong2 *__restrict__ Q, 
    ulonglong2 *__restrict__ K,
    half *__restrict__ V,
    float2 *output,
    float2 *sddmmResult);

__global__ void f3sKernel1tb1rwScheduled(
    const int *__restrict__ rowWindowOffset,
    const int *__restrict__ sortedRowWindows,
    const int *__restrict__ sparseAToXidx, 
    const uint64_t *__restrict__ tcbBitMap,
    int embeddingDim,
    int nRw,
    ulonglong2 *__restrict__ Q, 
    ulonglong2 *__restrict__ K, 
    half *__restrict__ V,
    float2 *output,
    float2 *sddmmResult);

__global__ void f2sKernel1tb1rw(
    const int *__restrict__ rowWindowOffset,
    const int *__restrict__ sparseAToXidx, 
    const uint64_t *__restrict__ tcbBitMap,
    int embeddingDim,
    ulonglong2 *__restrict__ Q, 
    ulonglong2 *__restrict__ K, 
    half *__restrict__ V,
    float2 *output,
    float2 *sddmmResult);

__global__ void sddmmKernel1tbnrw(
  const int *__restrict__ rowWindowOffset,
  const int *__restrict__ tbBoundaries,
  const int *__restrict__ tcbRowid,
  const int *__restrict__ sparseAToXidx, 
  const uint64_t *__restrict__ tcbBitMap,
  int embeddingDim,
  torch::Half *__restrict__ Q, 
  torch::Half *__restrict__ K, 
  float2 *output);

std::vector<torch::Tensor> 
f3sCuda1tb1tcb(
    torch::Tensor rowWindowOffset,
    torch::Tensor sparseAToXidx, 
    torch::Tensor tcbBitMap,
    int numNodes, 
    int embeddingDim,
    torch::Tensor Q, torch::Tensor K, torch::Tensor V, 
    bool applySoftmax,
    bool saveSddmmResult){
  int nBlockEmbeddingDim = (embeddingDim + BLK_N - 1) / BLK_N;
  int nWarpPerBlock =  (nBlockEmbeddingDim + 2 - 1) / 2;
  const int nRowWindow = rowWindowOffset.size(0) - 1;
  int paddedLength = nRowWindow * BLK_M;
  auto output = torch::zeros({paddedLength, embeddingDim}, torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));
  dim3 grid(nRowWindow, 1, 1);
  dim3 block(WARP_SIZE, nWarpPerBlock, 1);
  int nTcb = sparseAToXidx.size(0)/BLK_N;
	torch::Tensor sddmmResult = torch::zeros({nTcb*BLK_M*BLK_N}, torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));
  float* sddmmResultPtr = saveSddmmResult ? sddmmResult.data_ptr<float>() : nullptr;
  int fixedSharedSize = nWarpPerBlock * BLK_M * BLK_M * sizeof(float);
  int dynamicSharedSize = applySoftmax ? fixedSharedSize + 2 * BLK_M * sizeof(float) : fixedSharedSize;
  #if BLK_M == 16 && BLK_N == 8 && BLK_K == 16
  f3sKernel1tb1tcb<<<grid, block, dynamicSharedSize>>>(
    rowWindowOffset.data_ptr<int>(), 
    sparseAToXidx.data_ptr<int>(),
    tcbBitMap.data_ptr<uint64_t>(),
    numNodes, embeddingDim,
    Q.data_ptr<torch::Half>(), 
    K.data_ptr<torch::Half>(), 
    V.data_ptr<torch::Half>(),
    output.data_ptr<float>(),
    sddmmResultPtr,
    applySoftmax);
  #else
  printf("only m16n8k16 is supported\n");
  #endif

  // check for error
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
  hipDeviceSynchronize();
  // remove padding
  output = output.index(
      {torch::indexing::Slice(0, numNodes), torch::indexing::Slice()});
  return {output, sddmmResult};
}

std::vector<torch::Tensor> 
f3sCuda1tb1rw(
    torch::Tensor rowWindowOffset,
    torch::Tensor sparseAToXidx,
    torch::Tensor tcbBitMap,
    int nNodes,
    int embeddingDim,
    torch::Tensor Q, torch::Tensor K, torch::Tensor V,
    int nWarpPerBlock,
    bool applySoftmax){
  int nTcb = sparseAToXidx.size(0)/BLK_M;
  torch::Tensor sddmmResult = torch::zeros({nTcb*BLK_M*BLK_M}, torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA)); 
  int nRowWindow = rowWindowOffset.size(0) - 1;
  int paddedLength = nRowWindow * BLK_M; 
  auto output = torch::zeros({paddedLength, embeddingDim}, torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));
  int sharedSize = BLK_M * embeddingDim * sizeof(half); // Q
  sharedSize += nWarpPerBlock * BLK_M * BLK_N * sizeof(half); // E
  sharedSize += nWarpPerBlock * 2 * BLK_M * sizeof(float); // row_max, row_sum, old_max, old_sum
  sharedSize += BLK_M * embeddingDim * sizeof(float); // O_frag
  printf("sharedSize: %d\n", sharedSize);
  dim3 grid(nRowWindow, 1, 1);
  dim3 block(WARP_SIZE, nWarpPerBlock, 1);
  if(applySoftmax){
    f3sKernel1tb1rw<<<grid, block, sharedSize>>>(
      rowWindowOffset.data_ptr<int>(), 
      sparseAToXidx.data_ptr<int>(),
      tcbBitMap.data_ptr<uint64_t>(),
      embeddingDim,
      reinterpret_cast<ulonglong2*>(Q.data_ptr<torch::Half>()), 
      reinterpret_cast<ulonglong2*>(K.data_ptr<torch::Half>()), 
      reinterpret_cast<half*>(V.data_ptr<torch::Half>()),
      reinterpret_cast<float2*>(output.data_ptr<float>()),
      reinterpret_cast<float2*>(sddmmResult.data_ptr<float>()));
  }
  else{
    f2sKernel1tb1rw<<<grid, block, sharedSize>>>(
      rowWindowOffset.data_ptr<int>(), 
      sparseAToXidx.data_ptr<int>(),
      tcbBitMap.data_ptr<uint64_t>(),
      embeddingDim,
      reinterpret_cast<ulonglong2*>(Q.data_ptr<torch::Half>()), 
      reinterpret_cast<ulonglong2*>(K.data_ptr<torch::Half>()), 
      reinterpret_cast<half*>(V.data_ptr<torch::Half>()),
      reinterpret_cast<float2*>(output.data_ptr<float>()),
      reinterpret_cast<float2*>(sddmmResult.data_ptr<float>()));
  }
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
  // remove padding
  output = output.index(
      {torch::indexing::Slice(0, nNodes), torch::indexing::Slice()});
  return {output, sddmmResult};
}

std::vector<torch::Tensor> 
f3sCuda1tb1rwScheduled(
    torch::Tensor rowWindowOffset,
    torch::Tensor sortedRowWindows,
    torch::Tensor sparseAToXidx,
    torch::Tensor tcbBitMap,
    int nNodes,
    int embeddingDim,
    torch::Tensor Q, torch::Tensor K, torch::Tensor V,
    int nWarpPerBlock){
  int nTcb = sparseAToXidx.size(0)/BLK_M;
  torch::Tensor sddmmResult = torch::zeros({nTcb*BLK_M*BLK_M}, torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA)); 
  int nRowWindow = rowWindowOffset.size(0) - 1;
  int paddedLength = nRowWindow * BLK_M; 
  auto output = torch::zeros({paddedLength, embeddingDim}, torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));
  int sharedSize = BLK_M * embeddingDim * sizeof(half); // Q
  sharedSize += nWarpPerBlock * BLK_M * BLK_N * sizeof(half); // E
  sharedSize += nWarpPerBlock * 2 * BLK_M * sizeof(float); // row_max, row_sum, old_max, old_sum
  sharedSize += BLK_M * embeddingDim * sizeof(float); // O_frag
  printf("sharedSize: %d\n", sharedSize);
  dim3 grid(nRowWindow, 1, 1);
  dim3 block(WARP_SIZE, nWarpPerBlock, 1);
  f3sKernel1tb1rwScheduled<<<grid, block, sharedSize>>>(
    rowWindowOffset.data_ptr<int>(), 
    sortedRowWindows.data_ptr<int>(), 
    sparseAToXidx.data_ptr<int>(),
    tcbBitMap.data_ptr<uint64_t>(),
    embeddingDim,
    nRowWindow,
    reinterpret_cast<ulonglong2*>(Q.data_ptr<torch::Half>()), 
    reinterpret_cast<ulonglong2*>(K.data_ptr<torch::Half>()), 
    reinterpret_cast<half*>(V.data_ptr<torch::Half>()),
    reinterpret_cast<float2*>(output.data_ptr<float>()),
    reinterpret_cast<float2*>(sddmmResult.data_ptr<float>()));
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
  // remove padding
  output = output.index(
      {torch::indexing::Slice(0, nNodes), torch::indexing::Slice()});
  return {output, sddmmResult};
}

std::vector<torch::Tensor> 
sddmmCuda1tbnrw(
    torch::Tensor rowWindowOffset,
    torch::Tensor tbBoundaries,
    torch::Tensor tcbRowid,
    torch::Tensor sparseAToXidx,
    torch::Tensor tcbBitMap,
    int nNodes,
    int embeddingDim,
    torch::Tensor Q, torch::Tensor K,
    int nWarpPerBlock){
  int nRowWindow = tbBoundaries.size(0) - 1;
  int nTcb = sparseAToXidx.size(0)/BLK_N;
  torch::Tensor sddmmResult = torch::zeros({nTcb*BLK_M*BLK_N}, 
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));  
  dim3 grid(nRowWindow, 1, 1);
  dim3 block(WARP_SIZE, nWarpPerBlock, 1);
  sddmmKernel1tbnrw<<<grid, block>>>(
    rowWindowOffset.data_ptr<int>(), 
    tbBoundaries.data_ptr<int>(),
    tcbRowid.data_ptr<int>(),
    sparseAToXidx.data_ptr<int>(),
    tcbBitMap.data_ptr<uint64_t>(),
    embeddingDim,
    Q.data_ptr<torch::Half>(), 
    K.data_ptr<torch::Half>(), 
    reinterpret_cast<float2*>(sddmmResult.data_ptr<float>()));
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
  return {sddmmResult};
}



// #if defined(BLK_M) && defined(BLK_N) && defined(BLK_K) && \
//     BLK_M == 16 && BLK_N == 8 && BLK_K == 16
#define bid blockIdx.x
#define tid (threadIdx.x + threadIdx.y * blockDim.x)
__global__ void f3sKernel1tb1tcb(
		const int *__restrict__ rowWindowOffset, 		 // offset of each row window.
		const int *__restrict__ sparseAToXidx,     // colid of each TC block nonzero element.
    const uint64_t *__restrict__ tcbBitMap,
    const int numNodes,
    const int embeddingDim,
		torch::Half *__restrict__ Q, 
    torch::Half *__restrict__ K, 
    torch::Half *__restrict__ V,
		float *output,
		float *sddmmResult,
    bool applySoftmax){
  volatile int warpId = threadIdx.y;    // warp_index handling multi-dimension > 16.
  volatile int laneId = threadIdx.x; // lanid of each warp.

  uint64_t* K_uint64 = reinterpret_cast<uint64_t*>(K);

  // row_max, row_sum (size BLK_M each) for online-softmax,
  // then Q_frag, then O_frag
  extern __shared__ float dynShm[]; 
  for(int i = tid; i < BLK_M*2+blockDim.y*BLK_M*BLK_M; i += blockDim.x*blockDim.y){
  // for(int i = tid; i < BLK_M*2; i += blockDim.x*blockDim.y){
    dynShm[i] = 0.0f;
  }
 
  float* sum = dynShm + BLK_M*2;
  float O_frag[8] = {0};// spmm result
  uint32_t Q_frag[4] = {0};
  setQFrag(Q_frag, reinterpret_cast<uint64_t*>(Q), bid, warpId, laneId, numNodes, embeddingDim);
  // setQFragShm(dynShm, reinterpret_cast<uint64_t*>(Q), bid, warpId, laneId, numNodes, embeddingDim);

  /////////////////////////////////
  // main loop
  /////////////////////////////////
  volatile bool lastBlock = false;
  for (int tcbId = rowWindowOffset[bid]; tcbId < rowWindowOffset[bid + 1]; tcbId+=2) {
    if((rowWindowOffset[bid + 1] - rowWindowOffset[bid]) % 2 && tcbId == rowWindowOffset[bid + 1] - 1){
      lastBlock = true;
    }
    {// sddmm
      uint32_t B_frag[2];
      float D_frag[4];
      // uint32_t* Q_frag = reinterpret_cast<uint32_t*>(dynShm + BLK_M*2 + warpId*blockDim.x*4);
      int colIdx = (warpId * BLK_M)/4 + (laneId % 4); 
      for(int i = 0; i < 2; i++){
        if(!lastBlock || i == 0){
          // Initialize B_frag from K
          // Note I'm swapping rows of B_frag because we swapped the columns of A_frag(Q)
          // index in terms of half2, only affect rowIdx
          int rowIdx = sparseAToXidx[(tcbId+i) * BLK_N + laneId / 4]; 
          uint64_t val = K_uint64[rowIdx * embeddingDim/4 + colIdx];
          B_frag[0] = static_cast<uint32_t>(val & 0xFFFFFFFFull);
          B_frag[1] = static_cast<uint32_t>(val >> 32);
          HMMA16816(D_frag[0], D_frag[1], D_frag[2], D_frag[3], 
                    // Q_frag[laneId], Q_frag[blockDim.x + laneId], Q_frag[blockDim.x*2 + laneId], Q_frag[blockDim.x*3 + laneId], 
                    Q_frag[0], Q_frag[1], Q_frag[2], Q_frag[3], 
                    B_frag[0], B_frag[1], 
                    0.0f, 0.0f, 0.0f, 0.0f);
          storePartialSumShm(i, sum, D_frag, warpId, laneId);
        }
      }
      __syncthreads();
      addPartialSums(sum, tcbId, tid, blockDim.y, tcbBitMap, lastBlock);
    }
    __syncthreads();

    if(sddmmResult != nullptr){
      saveSddmmResult(sum, sddmmResult, tcbId, lastBlock);
    }

    {// softmax + spmm
      uint32_t S_frag[4];// softmax/sddmm result
      if(applySoftmax){
        float D_frag[4];
        float2* sumFloat2 = reinterpret_cast<float2*>(sum);
        for(int j = 0; j < 2; j++){// 2 8x8 blocks in each 16x8 block
          int sumOffset = j*BLK_N*BLK_N/2;
          for(int i = 0; i < 2; i++){// 2 16x8 blocks
            if(!lastBlock || i == 0){
              float2 temp = sumFloat2[i*BLK_M*BLK_N/2 + sumOffset + laneId];
              D_frag[i*2] = temp.x;
              D_frag[i*2 + 1] = temp.y;
            }
            else{
              D_frag[i*2] = 0.0f;
              D_frag[i*2 + 1] = 0.0f;
            }
          }
          //max of the 4 elements in the same row across 2 16x8 blocks
          //need every warp to do this because they will need it for the next computation
          float maxOld = dynShm[j*BLK_N + laneId/4];

          float max = fmaxf(
            fmaxf(fmaxf(D_frag[0], D_frag[1]), fmaxf(D_frag[2], D_frag[3])), maxOld);
          reduceMax(max, laneId);

          for(int i = 0; i < 4; i++){
            if(D_frag[i] != 0.0f){
              D_frag[i] = __expf(D_frag[i] - max);
            }
          }

          float expMaxDiff = __expf(maxOld - max);

          if(warpId == 0){
            float sum = D_frag[0] + D_frag[1] + D_frag[2] + D_frag[3];
            reduceSum(sum);
            if(laneId % 4 == 0){
              dynShm[BLK_M + j*BLK_N + laneId/4] = dynShm[BLK_M + j*BLK_N + laneId/4] * expMaxDiff + sum;
            }
          }

          O_frag[j*2]   = O_frag[j*2]   * expMaxDiff;
          O_frag[j*2+1] = O_frag[j*2+1] * expMaxDiff;
          O_frag[j*2+4] = O_frag[j*2+4] * expMaxDiff;
          O_frag[j*2+5] = O_frag[j*2+5] * expMaxDiff;
          // float* O_frag = dynShm + BLK_M*2 + blockDim.x*blockDim.y*4 + warpId*blockDim.x*8;
          // O_frag[blockDim.x*j*2 + laneId]     = O_frag[blockDim.x*j*2 + laneId]   * (expMaxDiff);
          // O_frag[blockDim.x*(j*2+1) + laneId] = O_frag[blockDim.x*(j*2+1) + laneId] * (expMaxDiff);
          // O_frag[blockDim.x*(j*2+4) + laneId] = O_frag[blockDim.x*(j*2+4) + laneId] * (expMaxDiff);
          // O_frag[blockDim.x*(j*2+5) + laneId] = O_frag[blockDim.x*(j*2+5) + laneId] * (expMaxDiff);

          if(warpId == 0 && laneId % 4 == 0){
            dynShm[j*BLK_N + laneId/4] = max;
          }

          Half2Uint32 h2U32Converter;
          for(int i = 0; i < 2; i++){
            h2U32Converter.h2.x = __float2half(D_frag[i*2]);
            h2U32Converter.h2.y = __float2half(D_frag[i*2+1]);
            S_frag[i*2 + j] = h2U32Converter.u32;
          }
        }
      }
      else{
        float2* sumFloat2 = reinterpret_cast<float2*>(sum);
        for(int i = 0; i < 2; i++){// 2 16x8 blocks
          int sumOffset = i*BLK_M*BLK_N/2;
          Half2Uint32 h2U32Converter;
          if(!lastBlock || i == 0){
            for(int j = 0; j < 2; j++){// 2 8x8 blocks in each 16x8 block
              float2 temp = sumFloat2[sumOffset + j*BLK_N*BLK_N/2 + laneId];
              h2U32Converter.h2.x = __float2half(temp.x);
              h2U32Converter.h2.y = __float2half(temp.y);
              S_frag[i*2+j] = h2U32Converter.u32;
            }
          }
          else{
            S_frag[i*2] = 0;
            S_frag[i*2+1] = 0;
          }
        }
      }
      __syncthreads();
      //reset sum to 0
      // for(int i = tid; i < BLK_M * BLK_N * 2; i += blockDim.x * blockDim.y){
      //   sum[i] = 0.0f;
      // }
      for(int i = laneId; i < BLK_M*BLK_M; i += blockDim.x){
        sum[warpId*BLK_M*BLK_M + i] = 0.0f;
      }
      /////////
      // SpMM
      /////////
      {
        uint32_t B_frag[2];
        Half2Uint32 h2U32Converter;
        half temp_V[2];
        // float* O_frag = dynShm + BLK_M*2 + blockDim.x*blockDim.y*4 + warpId*blockDim.x*8;
        for(int j = 0; j < 2; j++){// 2 16x8 blocks
          int colIdx = (warpId*2+j) * BLK_N + laneId/4;
          for(int i = 0; i < 2; i++){// 2 8x8 blocks in each 16x8 block
            if(!lastBlock || i == 0){
              for(int k = 0; k < 2; k++){// 2 halfs in each 8x8 block
                int rowIdx = sparseAToXidx[(tcbId+i) * BLK_N + (laneId%4)*2 + k];
                temp_V[k] = V[rowIdx * embeddingDim + colIdx];
              }
              h2U32Converter.h2 = __halves2half2(temp_V[0], temp_V[1]);
              B_frag[i] = h2U32Converter.u32;
            }
            else{
              B_frag[i] = 0;
            }
          }
          // HMMA16816(O_frag[blockDim.x*j*4 + laneId], O_frag[blockDim.x*(j*4+1) + laneId], O_frag[blockDim.x*(j*4+2) + laneId], O_frag[blockDim.x*(j*4+3) + laneId], 
          HMMA16816(O_frag[4*j], O_frag[4*j+1], O_frag[4*j+2], O_frag[4*j+3],
                    S_frag[0], S_frag[1], S_frag[2], S_frag[3], 
                    B_frag[0], B_frag[1], 
                 // O_frag[blockDim.x*4*j + laneId], O_frag[blockDim.x*(4*j+1) + laneId], O_frag[blockDim.x*(4*j+2) + laneId], O_frag[blockDim.x*(4*j+3) + laneId]);
                    O_frag[4*j], O_frag[4*j+1], O_frag[4*j+2], O_frag[4*j+3]);
        }
      }
    }
  }
  // float* O_frag = dynShm + BLK_M*2 + blockDim.x*blockDim.y*4 + warpId*blockDim.x*8;
  if(applySoftmax){
    for(int i = 0; i < 2; i++){
      float rowSum = dynShm[BLK_M + laneId/4 + i*BLK_N ];
      if(rowSum != 0.0f){
        // O_frag[blockDim.x*i*2 + laneId] = O_frag[blockDim.x*i*2 + laneId] * (1.0f/rowSum);
        // O_frag[blockDim.x*(i*2+1) + laneId] = O_frag[blockDim.x*(i*2+1) + laneId] * (1.0f/rowSum);
        // O_frag[blockDim.x*(i*2+4) + laneId] = O_frag[blockDim.x*(i*2+4) + laneId] * (1.0f/rowSum);
        // O_frag[blockDim.x*(i*2+5) + laneId] = O_frag[blockDim.x*(i*2+5) + laneId] * (1.0f/rowSum);
        O_frag[i*2] = O_frag[i*2] * (1.0f/rowSum);
        O_frag[i*2+1] = O_frag[i*2+1] * (1.0f/rowSum);
        O_frag[i*2+4] = O_frag[i*2+4] * (1.0f/rowSum);
        O_frag[i*2+5] = O_frag[i*2+5] * (1.0f/rowSum);
        
      }
    }
  }
  for(int j=0; j < 2; j++){// 2 8x8 blocks in each 16x8 block
    int rowIdx = bid * BLK_M + (laneId / 4) + j * BLK_M/2;
    for(int i =0; i < 2; i++){// 2 16x8 blocks
      int colIdx = (warpId * 2 + i) * BLK_N + (laneId % 4) * 2;
      // output[rowIdx * embeddingDim + colIdx] = O_frag[blockDim.x*(i*4 + j*2) + laneId];
      // output[rowIdx * embeddingDim + colIdx + 1] = O_frag[blockDim.x*(i*4 + j*2 + 1) + laneId]; 
      output[rowIdx * embeddingDim + colIdx] = O_frag[i*4 + j*2];
      output[rowIdx * embeddingDim + colIdx + 1] = O_frag[i*4 + j*2 + 1]; 
    }
  }
}

// load Q from HBM to register. Permute columns
__device__ void loadQFragPermuteCol(volatile uint32_t *Q_frag, uint64_t *Q, int embeddingDim, int rowIdx, int colIdx) {
    uint64_t val = Q[rowIdx * embeddingDim + colIdx];
    Q_frag[0] = static_cast<uint32_t>(val & 0xFFFFFFFFull);
    Q_frag[2] = static_cast<uint32_t>(val >> 32);
    val = Q[(rowIdx+8) * embeddingDim + colIdx];
    Q_frag[1] = static_cast<uint32_t>(val & 0xFFFFFFFFull);
    Q_frag[3] = static_cast<uint32_t>(val >> 32);
}

// ind is in 128b interval
// Assume Q is stored in row-major order. We divide Q into N 16x32 element block. 
__device__ void loadQHbm2Shm128b(uint64_t* qShm, ulonglong2* Q, int embeddingDim, int ind){
  int qWidth = embeddingDim/8;
  ulonglong2 val = Q[ind];
  ulonglong2 val2 = Q[ind + BLK_M/2 * qWidth];
  qWidth *= 2; //convert to 64b width
  ind *= 2; //convert to 64b index
  int rid = ind / qWidth;
  int cid = ind % qWidth;
  // 8 because 32*sizeof(half)/sizeof(uint64_t) = 8
  int blockCid = cid / 8;
  int cidInBlock = cid % 8;
  int blockOffset = blockCid*BLK_M*8 + rid*8 + cidInBlock;
  qShm[blockOffset] = val.x;
  qShm[blockOffset + 1] = val2.x;
  qShm[blockOffset + 64] = val.y;
  qShm[blockOffset + 65] = val2.y;
}

// Pair with loadQHbm2Shm128b. 
// This function has each warp read 1 16x16 block. 
//Not following the register layout in ptx doc but reordering it to match how K is loaded.
__device__ void loadQFragShm(uint64_t* Q_frag, uint64_t* dynShm, int ind, int laneId) {
  // 4 because BLK_M*sizeof(half)/sizeof(uint64_t) = 4
  int offset = ind*BLK_M*4 + laneId*2;
  Q_frag[0] = dynShm[offset];
  Q_frag[1] = dynShm[offset + 1];
}

__device__ void storeEFragShm(float* E_frag, uint32_t* dynShm) {
  Half2Uint32 h2U32Converter;
  for(int i = 0; i < 2; i++){
    h2U32Converter.h2.x = __float2half(E_frag[i*2]);
    h2U32Converter.h2.y = __float2half(E_frag[i*2+1]);
    dynShm[i*32] = h2U32Converter.u32;
  }
}
__device__ void loadEFragShm(uint32_t* E_frag, uint32_t* dynShm) {
  E_frag[0] = dynShm[0];
  E_frag[1] = dynShm[32];
  E_frag[2] = dynShm[64];
  E_frag[3] = dynShm[96];
}

__device__ void loadOFragShm(volatile float* O_frag, float* dynShm, float* mTilde) {
  O_frag[0] = dynShm[0] * mTilde[0];
  O_frag[1] = dynShm[32] * mTilde[0];
  O_frag[2] = dynShm[64] * mTilde[BLK_M/2];
  O_frag[3] = dynShm[96] * mTilde[BLK_M/2];
}

__device__ void storeOFragShm(volatile float* O_frag, float* dynShm) {
  dynShm[0] = O_frag[0];
  dynShm[32] = O_frag[1];
  dynShm[64] = O_frag[2];
  dynShm[96] = O_frag[3];
}

__device__ void printQFrag(uint64_t* Q_frag_uint64, int laneId) {
  if(threadIdx.y == 0){
    Half2Uint32 h2U32Converter0;
    Half2Uint32 h2U32Converter1;
    Half2Uint32 h2U32Converter2;
    Half2Uint32 h2U32Converter3;
    h2U32Converter0.u32 = static_cast<uint32_t>(Q_frag_uint64[0]& 0xFFFFFFFFull);
    h2U32Converter1.u32 = static_cast<uint32_t>(Q_frag_uint64[0] >> 32);
    h2U32Converter2.u32 = static_cast<uint32_t>(Q_frag_uint64[1]& 0xFFFFFFFFull);
    h2U32Converter3.u32 = static_cast<uint32_t>(Q_frag_uint64[1] >> 32);
    printf("laneId: %d, Q0: %f, Q1: %f, Q2: %f, Q3: %f, Q4: %f, Q5: %f, Q6: %f, Q7: %f\n", laneId, __half2float(h2U32Converter0.h2.x), __half2float(h2U32Converter0.h2.y), __half2float(h2U32Converter1.h2.x), __half2float(h2U32Converter1.h2.y), __half2float(h2U32Converter2.h2.x), __half2float(h2U32Converter2.h2.y), __half2float(h2U32Converter3.h2.x), __half2float(h2U32Converter3.h2.y));
  }
}

__device__ void printKFrag(ulonglong2 val, int laneId) {
  Half2Uint32 h2U32Converter0;
  Half2Uint32 h2U32Converter1;
  Half2Uint32 h2U32Converter2;
  Half2Uint32 h2U32Converter3;
  h2U32Converter0.u32 = static_cast<uint32_t>(val.x & 0xFFFFFFFFull);
  h2U32Converter1.u32 = static_cast<uint32_t>(val.x >> 32);
  h2U32Converter2.u32 = static_cast<uint32_t>(val.y & 0xFFFFFFFFull);
  h2U32Converter3.u32 = static_cast<uint32_t>(val.y >> 32);
  printf("warpId: %d, laneId: %d, K0: %f, K1: %f, K2: %f, K3: %f, K4: %f, K5: %f, K6: %f, K7: %f\n", threadIdx.y, laneId, __half2float(h2U32Converter0.h2.x), __half2float(h2U32Converter0.h2.y), __half2float(h2U32Converter1.h2.x), __half2float(h2U32Converter1.h2.y), __half2float(h2U32Converter2.h2.x), __half2float(h2U32Converter2.h2.y), __half2float(h2U32Converter3.h2.x), __half2float(h2U32Converter3.h2.y));
}

// Each warp computes 1 tcb of S.
// TODO: right now we need at least 2 warps per block because we need 2 tcbs to go to the spmm stage.
__global__ void f3sKernel1tb1rw(
    const int *__restrict__ rowWindowOffset,
    const int *__restrict__ sparseAToXidx, 
    const uint64_t *__restrict__ tcbBitMap,
    int embeddingDim,
    ulonglong2 *__restrict__ Q, 
    ulonglong2 *__restrict__ K, 
    half *__restrict__ V,
    float2 *output,
    float2 *sddmmResult) {
  volatile int laneId = threadIdx.x;
  int warpId = threadIdx.y;
  // contains a RW of Q
  extern __shared__ __align__(16) uint64_t dynShm1tb1rw[];
  __shared__ float maxOld[BLK_M];
  // r_b in Alg 1
  __shared__ float sumOld[BLK_M];
  __shared__ float mTilde[BLK_M];
  {
    //initialize everything to 0
    int oOffset = (embeddingDim+blockDim.y*BLK_N)*BLK_M/4 + 2*blockDim.y*BLK_M/2;
    for(int i = tid; i < oOffset + embeddingDim*BLK_M/2; i += blockDim.x*blockDim.y){
      dynShm1tb1rw[i] = 0;
    }
    for(int i = tid; i < BLK_M; i += blockDim.x*blockDim.y){
      maxOld[i] = 0.0f;
      sumOld[i] = 0.0f;
      mTilde[i] = 0.0f;
    }
  }
  //BLK_M/2 because each thread loads 2 128b elements
  for(int i = tid; i < (BLK_M/2)*embeddingDim/8; i += blockDim.x*blockDim.y){
    loadQHbm2Shm128b(dynShm1tb1rw, Q+bid*BLK_M*embeddingDim/8, embeddingDim, i);
  }
  __syncthreads();

  int niter = ((rowWindowOffset[bid+1] - rowWindowOffset[bid])*2 + blockDim.y - 1)/blockDim.y;
  #pragma unroll 1
  for(int iter = 0; iter < niter; iter++){
    int iterTcbStart = rowWindowOffset[bid] + iter*blockDim.y/2;
    // number of 16x16 blocks in S/E being computed in this iteration.
    int nBlock = min(blockDim.y/2, rowWindowOffset[bid+1]-iterTcbStart);
    float S_frag[4] = {0.0f};
    int warpTcbId = warpId/2 + iterTcbStart;
    if(warpId < nBlock*2){
      {//sddmm
        int kOffset = sparseAToXidx[warpTcbId*BLK_M + (warpId%2)*BLK_N + laneId/4] * embeddingDim/8 + laneId % 4;
        for(int i = 0; i < embeddingDim/BLK_K; i+=2) {
          //load K with permuted columns
          ulonglong2 val = K[kOffset + i*BLK_K/8];
          uint64_t Q_frag[2];
          loadQFragShm(Q_frag, dynShm1tb1rw, i, laneId);
          HMMA16816(S_frag[0], S_frag[1], S_frag[2], S_frag[3], 
                    static_cast<uint32_t>(Q_frag[0] & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(Q_frag[1] & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(Q_frag[0] >> 32), 
                    static_cast<uint32_t>(Q_frag[1] >> 32), 
                    static_cast<uint32_t>(val.x & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(val.x >> 32), 
                    S_frag[0], S_frag[1], S_frag[2], S_frag[3]);
          loadQFragShm(Q_frag, dynShm1tb1rw, i+1, laneId);
          HMMA16816(S_frag[0], S_frag[1], S_frag[2], S_frag[3], 
                    static_cast<uint32_t>(Q_frag[0] & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(Q_frag[1] & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(Q_frag[0] >> 32), 
                    static_cast<uint32_t>(Q_frag[1] >> 32), 
                    static_cast<uint32_t>(val.y & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(val.y >> 32), 
                    S_frag[0], S_frag[1], S_frag[2], S_frag[3]);
        }
        int bitIdx = 63 - laneId*2;
        for(int i = 0; i < 4; i++){
          uint64_t bitMask = 1ULL << (bitIdx - i%2);
          S_frag[i] = (tcbBitMap[warpTcbId*4+(warpId%2)*2+i/2] & bitMask) == 0 ? 0.0f : S_frag[i];
        }
      }
      // {//save sddmm result
      //   int offset = warpTcbId*BLK_M*BLK_M + (warpId%2)*BLK_M*BLK_N + laneId*2;
      //   for(int j = 0; j < 2; j++){ // 2 8x8 blocks in each 16x8 block
      //     int sumOffset = j*BLK_N*BLK_N;
      //     float2 val;
      //     val.x = S_frag[j*2];
      //     val.y = S_frag[j*2+1];
      //     sddmmResult[(offset + sumOffset)/2] = val;
      //   }
      // }
      {//online softmax
        float* maxPtr = reinterpret_cast<float*>(dynShm1tb1rw) + (embeddingDim + blockDim.y*BLK_N)*BLK_M/2 + laneId/4 + blockDim.y*BLK_M;
        //save max of each row within the warp to shared memory for cross-warp communication
        for(int i=0; i<2; i++){
          float localMax = fmaxf(S_frag[i*2], S_frag[i*2+1]);
          reduceMax(localMax, laneId);
          if(laneId % 4 == 0){
            maxPtr[warpId*BLK_M + i*BLK_M/2] = localMax;
          }
        }
      }
    }
    __syncthreads();
    if(warpId < nBlock*2){
      float* sumPtr = reinterpret_cast<float*>(dynShm1tb1rw) + (embeddingDim + blockDim.y*BLK_N)*BLK_M/2;
      for(int i=0; i<2; i++){
        int offset = i*BLK_M/2 + laneId/4;
        float* maxPtr = sumPtr + blockDim.y*BLK_M + offset;
        float newGlobalMax = maxOld[offset];
        //we have blockDim.y columns reserved for local sum of each warp
        //but only nBlock*2 are used
        for(int j=0; j<nBlock*2; j++){
          newGlobalMax = fmaxf(maxPtr[j*BLK_M], newGlobalMax);
        }
        if(warpId == 0 && laneId % 4 == 0){
          mTilde[offset] = __expf(maxOld[offset] - newGlobalMax);
          maxOld[offset] = newGlobalMax;
        }
        //compute E, ignore 0s
        S_frag[i*2] = S_frag[i*2]==0.0f ? 0.0f : __expf(S_frag[i*2] - newGlobalMax);
        S_frag[i*2+1] = S_frag[i*2+1]==0.0f ? 0.0f : __expf(S_frag[i*2+1] - newGlobalMax);
        //compute row sum and save to shared memory
        float localSum = S_frag[i*2] + S_frag[i*2+1];
        reduceSum(localSum);
        if(laneId % 4 == 0){
          sumPtr[warpId*BLK_M + offset] = localSum;
        }
      }
      int eOffset = (embeddingDim + warpId*BLK_N)*BLK_M/2+laneId;
      storeEFragShm(S_frag, reinterpret_cast<uint32_t*>(dynShm1tb1rw)+eOffset);
    }
    __syncthreads();
    //Could try moving this block to the end of the loop
    if(tid < BLK_M){
      float* sumPtr = reinterpret_cast<float*>(dynShm1tb1rw) + (embeddingDim + blockDim.y*BLK_N)*BLK_M/2 + laneId;
      //update r_b
      float rowSum = 0.0f;
      //we have blockDim.y columns reserved for local sum of each warp
      //but only nBlock*2 are used
      for(int j=0; j<nBlock*2; j++){
        rowSum += sumPtr[j*BLK_M];
      }
      sumOld[laneId] = fmaf(mTilde[laneId], sumOld[laneId], rowSum);
    }
    __syncthreads();
    {//SpMM
      int oOffset_base = (embeddingDim+blockDim.y*BLK_N)*BLK_M/2 + 2*blockDim.y*BLK_M + laneId;
      for(int i=warpId; i<embeddingDim/BLK_N; i+=blockDim.y){
        int oOffset = oOffset_base + i*BLK_M*BLK_N;
        volatile float O_frag[4];
        loadOFragShm(O_frag, reinterpret_cast<float*>(dynShm1tb1rw)+oOffset, mTilde+laneId/4);
        for(int j=0; j<nBlock; j++){
          uint32_t E_frag[4];
          uint32_t B_frag[2];
          //load E
          int eOffset = (embeddingDim+j*BLK_M)*BLK_M/2 + laneId;
          loadEFragShm(E_frag, reinterpret_cast<uint32_t*>(dynShm1tb1rw)+eOffset);
          //load V
          int sparseAToXidxOffset = (iterTcbStart+j)*BLK_M + (laneId%4)*2;
          for(int k = 0; k < 2; k++){
            sparseAToXidxOffset += k*BLK_N;
            Half2Uint32 h2U32Converter;
            int offset = sparseAToXidx[sparseAToXidxOffset]*embeddingDim+ i*BLK_N + laneId/4;
            h2U32Converter.h2.x = V[offset];
            offset = sparseAToXidx[sparseAToXidxOffset+1]*embeddingDim+ i*BLK_N + laneId/4;
            h2U32Converter.h2.y = V[offset];
            B_frag[k] = h2U32Converter.u32;
          }
          HMMA16816(O_frag[0], O_frag[1], O_frag[2], O_frag[3], 
                    E_frag[0], E_frag[1], E_frag[2], E_frag[3], 
                    B_frag[0], B_frag[1], 
                    O_frag[0], O_frag[1], O_frag[2], O_frag[3]);
        }
        storeOFragShm(O_frag, reinterpret_cast<float*>(dynShm1tb1rw)+oOffset);
      }
    }
  }
  __syncthreads();
  float invR0 = sumOld[laneId/4] == 0.0f ? 0.0f : 1.0f/sumOld[laneId/4];
  float invR1 = sumOld[BLK_M/2 + laneId/4] == 0.0f ? 0.0f : 1.0f/sumOld[BLK_M/2 + laneId/4];
  //points to (laneId)th element of O
  int oOffset = (embeddingDim+blockDim.y*BLK_N)*BLK_M/2 + 2*blockDim.y*BLK_M + laneId;
  //offset in terms of number of elements,
  //have to be divided by 2 to get the index of the float2
  int outputOffset = (bid*BLK_M + laneId/4)*embeddingDim + (laneId%4)*2;
  for(int i = warpId; i < embeddingDim/BLK_N; i += blockDim.y){
    int offset = oOffset + i*BLK_M*BLK_N;
    float* oPtr = reinterpret_cast<float*>(dynShm1tb1rw) + offset;
    float2 val;
    val.x = oPtr[0] * invR0;
    val.y = oPtr[32] * invR0;
    output[(outputOffset + i*BLK_N)/2] = val;
    val.x = oPtr[64] * invR1;
    val.y = oPtr[96] * invR1;
    output[(outputOffset + i*BLK_N + BLK_M/2*embeddingDim)/2] = val;
  }
}

__global__ void f3sKernel1tb1rwScheduled(
    const int *__restrict__ rowWindowOffset,
    const int *__restrict__ sortedRowWindows,
    const int *__restrict__ sparseAToXidx, 
    const uint64_t *__restrict__ tcbBitMap,
    int embeddingDim,
    int nRw,
    ulonglong2 *__restrict__ Q, 
    ulonglong2 *__restrict__ K, 
    half *__restrict__ V,
    float2 *output,
    float2 *sddmmResult) {
  Scheduler scheduler;
  volatile int laneId = threadIdx.x;
  int warpId = threadIdx.y;
  // contains a RW of Q
  extern __shared__ __align__(16) uint64_t dynShm1tb1rw[];
  __shared__ float maxOld[BLK_M];
  // r_b in Alg 1
  __shared__ float sumOld[BLK_M];
  __shared__ float mTilde[BLK_M];

  while(scheduler.next_iter(sortedRowWindows, nRw)){
    int niter = ((rowWindowOffset[scheduler.targetRw+1] - rowWindowOffset[scheduler.targetRw])*2
                 + blockDim.y - 1)/blockDim.y;
    {
      //initialize everything to 0
      int oOffset = (embeddingDim+blockDim.y*BLK_N)*BLK_M/4 + 2*blockDim.y*BLK_M/2;
      for(int i = tid; i < oOffset + embeddingDim*BLK_M/2; i += blockDim.x*blockDim.y){
        dynShm1tb1rw[i] = 0;
      }
      for(int i = tid; i < BLK_M; i += blockDim.x*blockDim.y){
        maxOld[i] = 0.0f;
        sumOld[i] = 0.0f;
        mTilde[i] = 0.0f;
      }
    }
    //BLK_M/2 because each thread loads 2 128b elements
    for(int i = tid; i < (BLK_M/2)*embeddingDim/8; i += blockDim.x*blockDim.y){
      loadQHbm2Shm128b(dynShm1tb1rw, Q+scheduler.targetRw*BLK_M*embeddingDim/8, embeddingDim, i);
    }
    __syncthreads();
    #pragma unroll 1
    for(int iter = 0; iter < niter; iter++){
      int iterTcbStart = rowWindowOffset[scheduler.targetRw] + iter*blockDim.y/2;
      // number of 16x16 blocks in S/E being computed in this iteration.
      int nBlock = min(blockDim.y/2, rowWindowOffset[scheduler.targetRw+1]-iterTcbStart);
      float S_frag[4] = {0.0f};
      int warpTcbId = warpId/2 + iterTcbStart;
      if(warpId < nBlock*2){
        {//sddmm
          int kOffset = sparseAToXidx[warpTcbId*BLK_M + (warpId%2)*BLK_N + laneId/4] 
                        * embeddingDim/8 + laneId % 4;
          for(int i = 0; i < embeddingDim/BLK_K; i+=2) {
            //load K with permuted columns
            ulonglong2 val = K[kOffset + i*BLK_K/8];
            uint64_t Q_frag[2];
            loadQFragShm(Q_frag, dynShm1tb1rw, i, laneId);
            HMMA16816(S_frag[0], S_frag[1], S_frag[2], S_frag[3], 
                      static_cast<uint32_t>(Q_frag[0] & 0xFFFFFFFFull), 
                      static_cast<uint32_t>(Q_frag[1] & 0xFFFFFFFFull), 
                      static_cast<uint32_t>(Q_frag[0] >> 32), 
                      static_cast<uint32_t>(Q_frag[1] >> 32), 
                      static_cast<uint32_t>(val.x & 0xFFFFFFFFull), 
                      static_cast<uint32_t>(val.x >> 32), 
                      S_frag[0], S_frag[1], S_frag[2], S_frag[3]);
            loadQFragShm(Q_frag, dynShm1tb1rw, i+1, laneId);
            HMMA16816(S_frag[0], S_frag[1], S_frag[2], S_frag[3], 
                      static_cast<uint32_t>(Q_frag[0] & 0xFFFFFFFFull), 
                      static_cast<uint32_t>(Q_frag[1] & 0xFFFFFFFFull), 
                      static_cast<uint32_t>(Q_frag[0] >> 32), 
                      static_cast<uint32_t>(Q_frag[1] >> 32), 
                      static_cast<uint32_t>(val.y & 0xFFFFFFFFull), 
                      static_cast<uint32_t>(val.y >> 32), 
                      S_frag[0], S_frag[1], S_frag[2], S_frag[3]);
          }
          int bitIdx = 63 - laneId*2;
          for(int i = 0; i < 4; i++){
            uint64_t bitMask = 1ULL << (bitIdx - i%2);
            S_frag[i] = (tcbBitMap[warpTcbId*4+(warpId%2)*2+i/2] & bitMask) == 0 ? 0.0f : S_frag[i];
          }
        }
        // {//save sddmm result
        //   int offset = warpTcbId*BLK_M*BLK_M + (warpId%2)*BLK_M*BLK_N + laneId*2;
        //   for(int j = 0; j < 2; j++){ // 2 8x8 blocks in each 16x8 block
        //     int sumOffset = j*BLK_N*BLK_N;
        //     float2 val;
        //     val.x = S_frag[j*2];
        //     val.y = S_frag[j*2+1];
        //     sddmmResult[(offset + sumOffset)/2] = val;
        //   }
        // }
        {//online softmax
          float* maxPtr = reinterpret_cast<float*>(dynShm1tb1rw) + (embeddingDim + blockDim.y*BLK_N)*BLK_M/2 + laneId/4 + blockDim.y*BLK_M;
          //save max of each row within the warp to shared memory for cross-warp communication
          for(int i=0; i<2; i++){
            float localMax = fmaxf(S_frag[i*2], S_frag[i*2+1]);
            reduceMax(localMax, laneId);
            if(laneId % 4 == 0){
              maxPtr[warpId*BLK_M + i*BLK_M/2] = localMax;
            }
          }
        }
      }
      __syncthreads();
      if(warpId < nBlock*2){
        float* sumPtr = reinterpret_cast<float*>(dynShm1tb1rw) + (embeddingDim + blockDim.y*BLK_N)*BLK_M/2;
        for(int i=0; i<2; i++){
          int offset = i*BLK_M/2 + laneId/4;
          float* maxPtr = sumPtr + blockDim.y*BLK_M + offset;
          float newGlobalMax = maxOld[offset];
          //we have blockDim.y columns reserved for local sum of each warp
          //but only nBlock*2 are used
          for(int j=0; j<nBlock*2; j++){
            newGlobalMax = fmaxf(maxPtr[j*BLK_M], newGlobalMax);
          }
          if(warpId == 0 && laneId % 4 == 0){
            mTilde[offset] = __expf(maxOld[offset] - newGlobalMax);
            maxOld[offset] = newGlobalMax;
          }
          //compute E, ignore 0s
          S_frag[i*2] = S_frag[i*2]==0.0f ? 0.0f : __expf(S_frag[i*2] - newGlobalMax);
          S_frag[i*2+1] = S_frag[i*2+1]==0.0f ? 0.0f : __expf(S_frag[i*2+1] - newGlobalMax);
          //compute row sum and save to shared memory
          float localSum = S_frag[i*2] + S_frag[i*2+1];
          reduceSum(localSum);
          if(laneId % 4 == 0){
            sumPtr[warpId*BLK_M + offset] = localSum;
          }
        }
        int eOffset = (embeddingDim + warpId*BLK_N)*BLK_M/2+laneId;
        storeEFragShm(S_frag, reinterpret_cast<uint32_t*>(dynShm1tb1rw)+eOffset);
      }
      __syncthreads();
      //Could try moving this block to the end of the loop
      if(tid < BLK_M){
        float* sumPtr = reinterpret_cast<float*>(dynShm1tb1rw) + (embeddingDim + blockDim.y*BLK_N)*BLK_M/2 + laneId;
        //update r_b
        float rowSum = 0.0f;
        //we have blockDim.y columns reserved for local sum of each warp
        //but only nBlock*2 are used
        for(int j=0; j<nBlock*2; j++){
          rowSum += sumPtr[j*BLK_M];
        }
        sumOld[laneId] = fmaf(mTilde[laneId], sumOld[laneId], rowSum);
      }
      __syncthreads();
      {//SpMM
        int oOffset_base = (embeddingDim+blockDim.y*BLK_N)*BLK_M/2 + 2*blockDim.y*BLK_M + laneId;
        for(int i=warpId; i<embeddingDim/BLK_N; i+=blockDim.y){
          int oOffset = oOffset_base + i*BLK_M*BLK_N;
          volatile float O_frag[4];
          loadOFragShm(O_frag, reinterpret_cast<float*>(dynShm1tb1rw)+oOffset, mTilde+laneId/4);
          for(int j=0; j<nBlock; j++){
            uint32_t E_frag[4];
            uint32_t B_frag[2];
            //load E
            int eOffset = (embeddingDim+j*BLK_M)*BLK_M/2 + laneId;
            loadEFragShm(E_frag, reinterpret_cast<uint32_t*>(dynShm1tb1rw)+eOffset);
            //load V
            int sparseAToXidxOffset = (iterTcbStart+j)*BLK_M + (laneId%4)*2;
            for(int k = 0; k < 2; k++){
              sparseAToXidxOffset += k*BLK_N;
              Half2Uint32 h2U32Converter;
              int offset = sparseAToXidx[sparseAToXidxOffset]*embeddingDim+ i*BLK_N + laneId/4;
              h2U32Converter.h2.x = V[offset];
              offset = sparseAToXidx[sparseAToXidxOffset+1]*embeddingDim+ i*BLK_N + laneId/4;
              h2U32Converter.h2.y = V[offset];
              B_frag[k] = h2U32Converter.u32;
            }
            HMMA16816(O_frag[0], O_frag[1], O_frag[2], O_frag[3], 
                      E_frag[0], E_frag[1], E_frag[2], E_frag[3], 
                      B_frag[0], B_frag[1], 
                      O_frag[0], O_frag[1], O_frag[2], O_frag[3]);
          }
          storeOFragShm(O_frag, reinterpret_cast<float*>(dynShm1tb1rw)+oOffset);
        }
      }
    }
    __syncthreads();
    float invR0 = sumOld[laneId/4] == 0.0f ? 0.0f : __frcp_rn(sumOld[laneId/4]);
    float invR1 = sumOld[BLK_M/2 + laneId/4] == 0.0f ? 0.0f : __frcp_rn(sumOld[BLK_M/2 + laneId/4]);
    //points to (laneId)th element of O
    int oOffset = (embeddingDim+blockDim.y*BLK_N)*BLK_M/2 + 2*blockDim.y*BLK_M + laneId;
    //offset in terms of number of elements,
    //have to be divided by 2 to get the index of the float2
    int outputOffset = (scheduler.targetRw*BLK_M + laneId/4)*embeddingDim + (laneId%4)*2;
    for(int i = warpId; i < embeddingDim/BLK_N; i += blockDim.y){
      int offset = oOffset + i*BLK_M*BLK_N;
      float* oPtr = reinterpret_cast<float*>(dynShm1tb1rw) + offset;
      float2 val;
      val.x = oPtr[0] * invR0;
      val.y = oPtr[32] * invR0;
      output[(outputOffset + i*BLK_N)/2] = val;
      val.x = oPtr[64] * invR1;
      val.y = oPtr[96] * invR1;
      output[(outputOffset + i*BLK_N + BLK_M/2*embeddingDim)/2] = val;
    }
  }
}

// Each warp computes 1 tcb of S.
// TODO: right now we need at least 2 warps per block because we need 2 tcbs to go to the spmm stage.
__global__ void f2sKernel1tb1rw(
    const int *__restrict__ rowWindowOffset,
    const int *__restrict__ sparseAToXidx, 
    const uint64_t *__restrict__ tcbBitMap,
    int embeddingDim,
    ulonglong2 *__restrict__ Q, 
    ulonglong2 *__restrict__ K, 
    half *__restrict__ V,
    float2 *output,
    float2 *sddmmResult) {
  volatile int laneId = threadIdx.x;
  int warpId = threadIdx.y;
  // contains a RW of Q
  extern __shared__ __align__(16) uint64_t dynShm1tb1rw[];
  __shared__ float mTilde[BLK_M];
  {
    //initialize everything to 0
    int oOffset = (embeddingDim+blockDim.y*BLK_N)*BLK_M/4 + 2*blockDim.y*BLK_M/2;
    for(int i = tid; i < oOffset + embeddingDim*BLK_M/2; i += blockDim.x*blockDim.y){
      dynShm1tb1rw[i] = 0;
    }
    for(int i = tid; i < BLK_M; i += blockDim.x*blockDim.y){
      mTilde[i] = 1.0f;
    }
  }
  //BLK_M/2 because each thread loads 2 128b elements
  for(int i = tid; i < (BLK_M/2)*embeddingDim/8; i += blockDim.x*blockDim.y){
    loadQHbm2Shm128b(dynShm1tb1rw, Q+bid*BLK_M*embeddingDim/8, embeddingDim, i);
  }
  __syncthreads();

  int niter = ((rowWindowOffset[bid+1] - rowWindowOffset[bid]) + (blockDim.y/2) - 1)/(blockDim.y/2);
  #pragma unroll 1
  for(int iter = 0; iter < niter; iter++){
    int iterTcbStart = rowWindowOffset[bid] + iter*(blockDim.y/2);
    // number of 16x16 blocks in S/E being computed in this iteration.
    // This is a check in case the last iteration is not full
    int nBlock = min(blockDim.y/2, rowWindowOffset[bid+1]-iterTcbStart);
    float S_frag[4] = {0.0f};
    int warpTcbId = warpId/2 + iterTcbStart;
    __syncthreads();//DEBUG
    if(warpTcbId < rowWindowOffset[bid+1]){
      {//sddmm
        int kOffset = sparseAToXidx[warpTcbId*BLK_M + (warpId%2)*BLK_N + laneId/4] * embeddingDim/8 + laneId % 4;
        for(int i = 0; i < embeddingDim/BLK_K; i+=2) {
          //load K with permuted columns
          ulonglong2 val = K[kOffset + i*BLK_K/8];
          uint64_t Q_frag[2];
          loadQFragShm(Q_frag, dynShm1tb1rw, i, laneId);
          HMMA16816(S_frag[0], S_frag[1], S_frag[2], S_frag[3], 
                    static_cast<uint32_t>(Q_frag[0] & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(Q_frag[1] & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(Q_frag[0] >> 32), 
                    static_cast<uint32_t>(Q_frag[1] >> 32), 
                    static_cast<uint32_t>(val.x & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(val.x >> 32), 
                    S_frag[0], S_frag[1], S_frag[2], S_frag[3]);
          loadQFragShm(Q_frag, dynShm1tb1rw, i+1, laneId);
          HMMA16816(S_frag[0], S_frag[1], S_frag[2], S_frag[3], 
                    static_cast<uint32_t>(Q_frag[0] & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(Q_frag[1] & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(Q_frag[0] >> 32), 
                    static_cast<uint32_t>(Q_frag[1] >> 32), 
                    static_cast<uint32_t>(val.y & 0xFFFFFFFFull), 
                    static_cast<uint32_t>(val.y >> 32), 
                    S_frag[0], S_frag[1], S_frag[2], S_frag[3]);
        }
        int bitIdx = 63 - laneId*2;
        for(int i = 0; i < 4; i++){
          uint64_t bitMask = 1ULL << (bitIdx - i%2);
          S_frag[i] = (tcbBitMap[warpTcbId*4+(warpId%2)*2+i/2] & bitMask) == 0 ? 0.0f : S_frag[i];
        }
      }
      {//save sddmm result
        int offset = warpTcbId*BLK_M*BLK_M + (warpId%2)*BLK_M*BLK_N + laneId*2;
        for(int j = 0; j < 2; j++){ // 2 8x8 blocks in each 16x8 block
          int sumOffset = j*BLK_N*BLK_N;
          float2 val;
          val.x = S_frag[j*2];
          val.y = S_frag[j*2+1];
          sddmmResult[(offset + sumOffset)/2] = val;
        }
      }
    }
    __syncthreads();
    if(warpTcbId < rowWindowOffset[bid+1]){
      int eOffset = (embeddingDim + warpId*BLK_N)*BLK_M/2+laneId;
      storeEFragShm(S_frag, reinterpret_cast<uint32_t*>(dynShm1tb1rw)+eOffset);
    }
    __syncthreads();
    {//SpMM
      int oOffset_base = (embeddingDim+blockDim.y*BLK_N)*BLK_M/2 + 2*blockDim.y*BLK_M + laneId;
      for(int i=warpId; i<embeddingDim/BLK_N; i+=blockDim.y){
        int oOffset = oOffset_base + i*BLK_M*BLK_N;
        float O_frag[4];
        loadOFragShm(O_frag, reinterpret_cast<float*>(dynShm1tb1rw)+oOffset, mTilde+laneId/4);
        for(int j=0; j<nBlock; j++){
          uint32_t E_frag[4];
          uint32_t B_frag[2];
          //load E
          int eOffset = (embeddingDim+j*BLK_M)*BLK_M/2 + laneId;
          loadEFragShm(E_frag, reinterpret_cast<uint32_t*>(dynShm1tb1rw)+eOffset);
          //load V
          int sparseAToXidxOffset = (iterTcbStart+j)*BLK_M + (laneId%4)*2;
          for(int k = 0; k < 2; k++){
            sparseAToXidxOffset += k*BLK_N;
            Half2Uint32 h2U32Converter;
            int offset = sparseAToXidx[sparseAToXidxOffset]*embeddingDim+ i*BLK_N + laneId/4;
            h2U32Converter.h2.x = V[offset];
            offset = sparseAToXidx[sparseAToXidxOffset+1]*embeddingDim+ i*BLK_N + laneId/4;
            h2U32Converter.h2.y = V[offset];
            B_frag[k] = h2U32Converter.u32;
          }
          HMMA16816(O_frag[0], O_frag[1], O_frag[2], O_frag[3], 
                    E_frag[0], E_frag[1], E_frag[2], E_frag[3], 
                    B_frag[0], B_frag[1], 
                    O_frag[0], O_frag[1], O_frag[2], O_frag[3]);
        }
        storeOFragShm(O_frag, reinterpret_cast<float*>(dynShm1tb1rw)+oOffset);
      }
    }
    __syncthreads();
  }
  __syncthreads();
  //points to (laneId)th element of O
  int oOffset = (embeddingDim+blockDim.y*BLK_N)*BLK_M/2 + 2*blockDim.y*BLK_M + laneId;
  //offset in terms of number of elements,
  //have to be divided by 2 to get the index of the float2
  int outputOffset = (bid*BLK_M + laneId/4)*embeddingDim + (laneId%4)*2;
  for(int i = warpId; i < embeddingDim/BLK_N; i += blockDim.y){
    int offset = oOffset + i*BLK_M*BLK_N;
    float* oPtr = reinterpret_cast<float*>(dynShm1tb1rw) + offset;
    float2 val;
    val.x = oPtr[0];
    val.y = oPtr[32];
    output[(outputOffset + i*BLK_N)/2] = val;
    val.x = oPtr[64];
    val.y = oPtr[96];
    output[(outputOffset + i*BLK_N + BLK_M/2*embeddingDim)/2] = val;
  }
}

// Each tb computes multiple row windows of S
__global__ void sddmmKernel1tbnrw(
    const int *__restrict__ rowWindowOffset,
    const int *__restrict__ tbBoundaries,
    const int *__restrict__ tcbRowId,
    const int *__restrict__ sparseAToXidx, 
    const uint64_t *__restrict__ tcbBitMap,
    int embeddingDim,
    torch::Half *__restrict__ Q, 
    torch::Half *__restrict__ K, 
    float2 *output) {
  volatile int laneId = threadIdx.x;
  int warpId = threadIdx.y;
  volatile int tidInGroup = laneId % 4;
  int tcbStart = rowWindowOffset[tbBoundaries[bid]];
  int tcbEnd = rowWindowOffset[tbBoundaries[bid+1]];
  for(int tcbId = tcbStart + warpId; tcbId < tcbEnd; tcbId += blockDim.y) {
    volatile float S_frag[4] = {0.0f};
    volatile uint32_t Q_frag[4];
    volatile uint32_t K_frag[2];
    int rowIdxQ = tcbRowId[tcbId]*BLK_M + laneId/4;
    int rowIdxK = sparseAToXidx[tcbId*BLK_N + laneId/4] * embeddingDim/4; 
    for(int i = 0; i < embeddingDim/BLK_K; i++) {
      loadQFragPermuteCol(Q_frag, reinterpret_cast<uint64_t*>(Q), embeddingDim/4, rowIdxQ, i*BLK_K/4 + tidInGroup);
      uint64_t val = reinterpret_cast<uint64_t*>(K)[rowIdxK + i*BLK_K/4 + tidInGroup];
      K_frag[0] = static_cast<uint32_t>(val & 0xFFFFFFFFull);
      K_frag[1] = static_cast<uint32_t>(val >> 32);
      HMMA16816(S_frag[0], S_frag[1], S_frag[2], S_frag[3], 
                Q_frag[0], Q_frag[1], Q_frag[2], Q_frag[3], 
                K_frag[0], K_frag[1], 
                S_frag[0], S_frag[1], S_frag[2], S_frag[3]);
    }
    int bitIdx = 63 - laneId*2;
    for(int i = 0; i < 4; i++){
      uint64_t bitMask = 1ULL << (bitIdx - i%2);
      S_frag[i] = (tcbBitMap[tcbId*2+i/2] & bitMask) == 0 ? 0.0f : S_frag[i];
    }

    int offset = tcbId * BLK_M * BLK_N;
    for(int j = 0; j < 2; j++){ // 2 8x8 blocks in each 16x8 block
      int sumOffset = j*BLK_N*BLK_N + laneId*2;
      float2 val;
      val.x = S_frag[j*2];
      val.y = S_frag[j*2+1];
      output[(offset + sumOffset)/2] = val;
    }
  }
}
// #endif